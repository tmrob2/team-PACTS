#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>

struct CSRMatrix {
    int nzmax;
    int m;
    int n; 
    int *p;
    int *i;
    int *x;
    int nz;
};

void value_iteration(struct CSRMatrix x[], size_t len) {
    // convert each of the CSR matrices into a sparse matrix
    // TODO need a pointer for the returns from value iteration
}


extern "C" {

void test_csr_to_cuda_spcsr(struct CSRMatrix x[], size_t len) {
    hipsparseHandle_t handle = NULL;
    hipsparseCreate(&handle);
    hipsparseSpMatDescr_t descrA = NULL;

    hipsparseStatus_t status = HIPSPARSE_STATUS_SUCCESS;
}

}