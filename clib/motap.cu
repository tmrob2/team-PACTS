#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include "hipblas.h"
#include "thrust/device_vector.h"
#include "thrust/device_ptr.h"
#include <thrust/extrema.h>
#include <thrust/execution_policy.h>

/*
#######################################################################
#                           KERNELS                                   #
#######################################################################
*/


extern "C" {

__global__ void abs_diff(float *a, float *b, float *c, int m) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x; // HANDLE THE DATA AT THIS INDEX
    if (tid < m) {
        // compute the absolute diff between two elems
        float temp = fabsf(a[tid] - b[tid]);
        c[tid] = temp;
    } 
}


/*
#######################################################################
#                           C HELPER                                  #
#######################################################################
*/

struct CSparse {
    int nz;
    int m;
    int n;
    int *p;
    int *i;
    float *x;
};

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("cuSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUBLAS(func)                                                     \
{                                                                              \
    hipblasStatus_t status = (func);                                            \
    if (status != HIPBLAS_STATUS_SUCCESS) {                                     \
        printf("CUBLAS API failed at line %d with error: %d\n",                \
               __LINE__, status);                                              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

/*
#######################################################################
#                              CUDA                                   #
#######################################################################
*/


void test_csr_spmv(
    int *csr_row, 
    int *csr_col, 
    float *csr_vals, 
    float *x,
    float *y,
    int nnz, 
    int sizeof_row, 
    int m, 
    int n
    ) {
    // sizeof_row is the size of csr_row
    // sizeof_col is the size of csr_col
    // m number of rows in the matrix
    // n number of cols in the matrix
    // nnz is the size of the csr_vals
    // create a sparse handle
    hipsparseHandle_t handle = NULL;
    hipsparseCreate(&handle);
    hipsparseSpMatDescr_t descrC = NULL;
    hipsparseStatus_t status = HIPSPARSE_STATUS_SUCCESS;

    //allocate dCsrRowPtr, dCsrColPtr, dCsrValPtr
    int *dCsrRowPtr, *dCsrColPtr;
    float *dCsrValPtr;

    // allocate device memory to store the sparse CSR
    hipMalloc((void **)&dCsrValPtr, sizeof(float) * nnz);
    hipMalloc((void **)&dCsrColPtr, sizeof(int) * nnz);
    hipMalloc((void **)&dCsrRowPtr, sizeof(int) * m);

    // Free the device memory allocated to the coo ptrs once they
    // the conversion from coo to csr has been completed
    hipMemcpy(dCsrValPtr, csr_vals, sizeof(float) * nnz, hipMemcpyHostToDevice);
    hipMemcpy(dCsrColPtr, csr_col, sizeof(int) * nnz, hipMemcpyHostToDevice);
    hipMemcpy(dCsrRowPtr, csr_row, sizeof(int) * m, hipMemcpyHostToDevice);

    // create the sparse CSR matrix in device memory
    status = hipsparseCreateCsr(
        &descrC, // MATRIX DESCRIPTION
        m, // NUMBER OF ROWS
        n, // NUMBER OF COLS
        nnz, // NUMBER OF NON ZERO VALUES
        dCsrRowPtr, // ROWS OFFSETS
        dCsrColPtr, // COL INDICES
        dCsrValPtr, // VALUES
        HIPSPARSE_INDEX_32I, // INDEX TYPE ROWS
        HIPSPARSE_INDEX_32I, // INDEX TYPE COLS
        HIPSPARSE_INDEX_BASE_ZERO, // BASE INDEX TYPE
        HIP_R_32F // DATA TYPE
    );

    float alpha = 1.0;
    float beta = 1.0;

    // assign the cuda memory for the vectors
    hipsparseDnVecDescr_t vecX, vecY;
    float *dX, *dY;
    void* dBuffer = NULL;
    size_t bufferSize = 0;
    hipMalloc((void**)&dX, n * sizeof(float));
    hipMalloc((void**)&dY, m * sizeof(float));

    // copy the vector from host memory to device memory
    hipMemcpy(dX, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dY, y, m * sizeof(float), hipMemcpyHostToDevice);

    // create a dense vector on device memory
    hipsparseCreateDnVec(&vecX, n, dX, HIP_R_32F);
    hipsparseCreateDnVec(&vecY, m, dY, HIP_R_32F);

    hipsparseSpMV_bufferSize(
        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, descrC, vecX, &beta, vecY, HIP_R_32F,
        HIPSPARSE_MV_ALG_DEFAULT, &bufferSize);
    hipMalloc(&dBuffer, bufferSize);

    hipsparseSpMV(
        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
        &alpha, descrC, vecX, &beta, vecY, HIP_R_32F, 
        HIPSPARSE_MV_ALG_DEFAULT, dBuffer);

    // Any algorithms get inserted here

    hipMemcpy(y, dY, m *sizeof(float), hipMemcpyDeviceToHost);

    //destroy the vector descriptors
    hipsparseDestroySpMat(descrC);
    hipsparseDestroyDnVec(vecX);
    hipsparseDestroyDnVec(vecY);
    hipsparseDestroy(handle);

    // Free the device memory
    hipFree(dCsrColPtr);
    hipFree(dCsrRowPtr);
    hipFree(dCsrValPtr);
    hipFree(dX);
    hipFree(dY);
    hipFree(dBuffer);

}

void test_csr_create(
    int m,
    int n,
    int nnz,
    int *i,
    int *j,
    int *x
) {
    hipsparseHandle_t handle = NULL;
    hipsparseCreate(&handle);

    hipsparseSpMatDescr_t descrP = NULL;
    hipsparseStatus_t status = HIPSPARSE_STATUS_SUCCESS;

    // allocated the device memory for the COO matrix
    int *dCOORowPtr, *dCOOColPtr;
    float *dCOOValPtr;

    // allocate device memory to store the sparse CSR
    hipMalloc((void **)&dCOOValPtr, sizeof(float) * nnz);
    hipMalloc((void **)&dCOORowPtr, sizeof(int) * nnz);
    hipMalloc((void **)&dCOOColPtr, sizeof(int) * nnz);

    hipMemcpy(dCOOValPtr, i, sizeof(float) * nnz, hipMemcpyHostToDevice);
    hipMemcpy(dCOOColPtr, j, sizeof(int) * nnz, hipMemcpyHostToDevice);
    hipMemcpy(dCOORowPtr, x, sizeof(int) * (m + 1), hipMemcpyHostToDevice);

    status = hipsparseCreateCsr(
        &descrP,
        m,
        n,
        nnz,
        dCOORowPtr,
        dCOOColPtr,
        dCOOValPtr,
        HIPSPARSE_INDEX_32I, // ROW OFFSET
        HIPSPARSE_INDEX_32I, // COL IND
        HIPSPARSE_INDEX_BASE_ZERO, // BASE INDEX TYPE
        HIP_R_32F // DATA TYPE
    );

    // convert the trans output to device memory

    hipsparseDestroySpMat(descrP);
    hipsparseDestroy(handle);

    // Free the device memory
    
    hipFree(dCOORowPtr);
    hipFree(dCOOColPtr);
    hipFree(dCOOValPtr);
}

int test_initial_policy_value(
    float *value,
    int pm,
    int pn,
    int pnz,
    int * pi,
    int * pj,
    float * px,
    int rm,
    int rn,
    int rnz,
    int *ri,
    int *rj,
    float *rx,
    float *x,
    float *y,
    float *w,
    float *rmv
    ) {
    /* 
    this test is to understand moving data onto CUDA so that
    a spmv can be performed with cublas, cusparse
    then a resulting sum ax + by

    Get the COO matrix into sparsescoo fmt

    Then multiply the COO by the initial value vector

    The rewards matrix is also sparse so it will need a sparse matrix descr
    as well. Multiply R by a repeated weight vector in the number 
    of prods and actions

    Finally sum the result

    This should happen in a loop until convergence

    I also want to do some wall timing to see some statistics on 
    the GPU 
    */
    //int *trans_output, *reward_output;
    //trans_output = (int *)malloc(block_size * max_state_space * sizeof(int));
    //reward_output = (int *)malloc(block_size * num_objectives * sizeof(int));

    // lets build the sparse transition matrix first

    hipsparseHandle_t handle = NULL;
    hipsparseCreate(&handle);
    hipError_t cudaStat;
    hipblasHandle_t blashandle;
    hipblasCreate(&blashandle);


    hipsparseSpMatDescr_t descrP = NULL;
    hipsparseSpMatDescr_t descrR = NULL;
    hipsparseStatus_t status = HIPSPARSE_STATUS_SUCCESS;
    hipblasStatus_t blas_status = HIPBLAS_STATUS_SUCCESS;

    // allocated the device memory for the COO matrix

    // ----------------------------------------------------------------
    //                       Transition Matrix
    // ----------------------------------------------------------------

    //allocate dCsrRowPtr, dCsrColPtr, dCsrValPtr
    int *dPCsrRowPtr, *dPCsrColPtr;
    float *dPCsrValPtr;

    // allocate device memory to store the sparse CSR 
    hipMalloc((void **)&dPCsrValPtr, sizeof(float) * pnz);
    hipMalloc((void **)&dPCsrRowPtr, sizeof(int) * pm);
    hipMalloc((void **)&dPCsrColPtr, sizeof(int) * pnz);

    hipMemcpy(dPCsrValPtr, px, sizeof(float) * pnz, hipMemcpyHostToDevice);
    hipMemcpy(dPCsrColPtr, pj, sizeof(int) * pnz, hipMemcpyHostToDevice);
    hipMemcpy(dPCsrRowPtr, pi, sizeof(int) * pm, hipMemcpyHostToDevice);
    
    // create the sparse CSR matrix in device memory
    status = hipsparseCreateCsr(
        &descrP, // MATRIX DESCRIPTION
        pm, // NUMBER OF ROWS
        pn, // NUMBER OF COLS
        pnz, // NUMBER OF NON ZERO VALUES
        dPCsrRowPtr, // ROWS OFFSETS
        dPCsrColPtr, // COL INDICES
        dPCsrValPtr, // VALUES
        HIPSPARSE_INDEX_32I, // INDEX TYPE ROWS
        HIPSPARSE_INDEX_32I, // INDEX TYPE COLS
        HIPSPARSE_INDEX_BASE_ZERO, // BASE INDEX TYPE
        HIP_R_32F // DATA TYPE
    );
    
    // ----------------------------------------------------------------
    //                       Rewards Matrix
    // ----------------------------------------------------------------
    
    int *dRCsrRowPtr, *dRCsrColPtr;
    float *dRCsrValPtr;

    // allocate device memory to store the sparse CSR 
    hipMalloc((void **)&dRCsrValPtr, sizeof(float) * rnz);
    hipMalloc((void **)&dRCsrRowPtr, sizeof(int) * rm);
    hipMalloc((void **)&dRCsrColPtr, sizeof(int) * rnz);
    printf("PRINTING COPIED REWARDS DATA\n");
    for (int k = 0; k < rnz; k ++) {
        printf("%f, ", rx[k]);
    }
    printf("\n");

    hipMemcpy(dRCsrValPtr, rx, sizeof(float) * rnz, hipMemcpyHostToDevice);
    hipMemcpy(dRCsrColPtr, rj, sizeof(int) * rnz, hipMemcpyHostToDevice);
    hipMemcpy(dRCsrRowPtr, ri, sizeof(int) * rm, hipMemcpyHostToDevice);

    // create the sparse CSR matrix in device memory
    printf("ROWS: %i, COLS: %i, NNZ: %i\n", rm, rn, rnz);
    status = hipsparseCreateCsr(
        &descrR, // MATRIX DESCRIPTION
        rm, // NUMBER OF ROWS
        rn, // NUMBER OF COLS
        rnz, // NUMBER OF NON ZERO VALUES
        dRCsrRowPtr, // ROWS OFFSETS
        dRCsrColPtr, // COL INDICES
        dRCsrValPtr, // VALUES
        HIPSPARSE_INDEX_32I, // INDEX TYPE ROWS
        HIPSPARSE_INDEX_32I, // INDEX TYPE COLS
        HIPSPARSE_INDEX_BASE_ZERO, // BASE INDEX TYPE
        HIP_R_32F // DATA TYPE
    );

    // ----------------------------------------------------------------
    //                      Start of VI
    // ----------------------------------------------------------------

    // --------------TRANSITION MATRIX MULTIPLICATION SETUP------------
    
    float alpha = 1.0;
    float beta = 1.0;
    float *epsilon = (float*) malloc(pm * sizeof(float));
    int iepsilon;

    // assign the cuda memory for the vectors
    hipsparseDnVecDescr_t vecX, vecY;
    int *d_arg_epsilon;
    float *dX, *dY, *d_tmp, *dZ, *dStaticY, *dOutput;
    void* dBuffer = NULL;
    size_t bufferSize = 0;

    hipMalloc((void**)&dX, pm * sizeof(float));
    hipMalloc((void**)&dOutput, pm * sizeof(float));
    hipMalloc((void**)&dY, pm * sizeof(float));
    hipMalloc((void**)&dZ, pm * sizeof(float));
    hipMalloc((void**)&dStaticY, pm * sizeof(float));
    hipMalloc((void**)&d_tmp, pm * sizeof(float));
    hipMalloc((void**)&d_arg_epsilon, sizeof(int));

    // create a initial Y vector
    float *static_y = (float*) calloc(pm, sizeof(float));
    
    // copy the vector from host memory to device memory
    hipMemcpy(dX, x, pn * sizeof(float), hipMemcpyHostToDevice);
    //hipMemcpy(dStaticY, static_y, pm * sizeof(float), hipMemcpyHostToDevice);

    // create a dense vector on device memory
    hipsparseCreateDnVec(&vecX, pn, dX, HIP_R_32F);
    hipsparseCreateDnVec(&vecY, pm, dY, HIP_R_32F);

    hipsparseSpMV_bufferSize(
        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, descrP, vecX, &beta, vecY, HIP_R_32F,
        HIPSPARSE_MV_ALG_DEFAULT, &bufferSize);
    hipMalloc(&dBuffer, bufferSize);
    
    // --------------REWARDS MATRIX MULTIPLICATION SETUP---------------

    float alphaR = 1.0;
    float betaR = 1.0;

    // assign the cuda memory for the vectors
    hipsparseDnVecDescr_t vecW, vecRMv;
    float *dRw, *dRMv, *dRstaticMx;
    void* dBufferR = NULL;
    size_t bufferSizeR = 0;

    //float *rmv = (float*) calloc(rm, sizeof(float));

    hipMalloc((void**)&dRw, rn * sizeof(float));
    hipMalloc((void**)&dRMv, rm * sizeof(float));
    hipMalloc((void**)&dRstaticMx, rm * sizeof(float));

    // copy the vector from host memory to device memory
    hipMemcpy(dRw, w, rn * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dRMv, rmv, rm * sizeof(float), hipMemcpyHostToDevice);
    //hipMemcpy(dRstaticMx, rmv, rm * sizeof(float), hipMemcpyHostToDevice);

    // create a dense vector on device memory
    hipsparseCreateDnVec(&vecW, rn, dRw, HIP_R_32F);
    hipsparseCreateDnVec(&vecRMv, rm, dRMv, HIP_R_32F);

    hipsparseSpMV_bufferSize(
        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alphaR, descrR, vecW, &betaR, vecRMv, HIP_R_32F,
        HIPSPARSE_MV_ALG_DEFAULT, &bufferSizeR);
    hipMalloc(&dBufferR, bufferSizeR);

    // ALGORITHM LOOP

    // Copy the zero vector to initialise Y -> captures A.x result 
    // for transition matrix
    //csparseDnVecSetValues(vecY, dY);
    //hipblasScopy(blashandle, pm, dYStatic, 1, dY, 1);
    // copy the static Y vector to initialise Y
    CHECK_CUSPARSE(hipsparseSpMV(
        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
        &alphaR, descrR, vecW, &betaR, vecRMv, HIP_R_32F, 
        HIPSPARSE_MV_ALG_DEFAULT, dBufferR));

    CHECK_CUDA(hipMemcpy(y, dRMv, pm *sizeof(float), hipMemcpyDeviceToHost));
    printf("PRINTING REWARDS VECTOR AFTER MxV\n");
    for (int k = 0; k < pm; k++) {
        printf("%f, ", y[k]);
    }
    printf("\n");

    for (int algo_i = 0; algo_i < 10; algo_i ++) {

        CHECK_CUSPARSE(hipsparseSpMV(
        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
        &alpha, descrP, vecX, &beta, vecY, HIP_R_32F, 
        HIPSPARSE_MV_ALG_DEFAULT, dBuffer));

        // push this into the algorithm loop


        // ---------------------SUM DENSE VECTORS-------------------------

        /* 
        The gpu memory shoulf already be allocated, i.e. we are summing
        dY + dRMv
        */
        hipblasSaxpy(blashandle, pm, &alpha, dRMv, 1, dY, 1);
        
        // ---------------------COMPUTE EPSILON---------------------------

        // what is the difference between dY and dX

        // EPSILON COMPUTATION
        abs_diff<<<pm,1>>>(dX, dY, dZ, pm);
        CHECK_CUDA(hipMemcpy(y, dY, pm *sizeof(float), hipMemcpyDeviceToHost));
        /*
        for (int k = 0; k < pm; k++) {
            printf("%.1f, ", y[k]);
        }
        printf("\n");
        */
        CHECK_CUBLAS(hipblasIsamax(blashandle, pm, dZ, 1, &iepsilon));
        CHECK_CUDA(hipMemcpy(epsilon, dZ, pm *sizeof(float), hipMemcpyDeviceToHost));
        //thrust::device_ptr<float> dev_ptr(dZ);
        //epsilon = dev_ptr[iepsilon];
        //epsilon = y[iepsilon];
        CHECK_CUBLAS(hipblasScopy(blashandle, pm, dY, 1, dX, 1));
        // RESET Y
        CHECK_CUBLAS(hipblasScopy(blashandle, pm, dStaticY, 1, dY, 1));
        // RESET RMV
        
        //CHECK_CUSPARSE(hipsparseDnVecSetValues(vecX, dX));
        //CHECK_CUSPARSE(hipsparseDnVecSetValues(vecY, dY));
    }
    
    
    //hipMemcpy(rmv, dRMv, rm *sizeof(float), hipMemcpyDeviceToHost);
    //destroy the vector descriptors
    hipsparseDestroySpMat(descrP);
    hipsparseDestroySpMat(descrR);
    hipsparseDestroyDnVec(vecX);
    hipsparseDestroyDnVec(vecY);
    hipsparseDestroyDnVec(vecRMv);
    hipsparseDestroyDnVec(vecW);
    hipsparseDestroy(handle);
    hipblasDestroy(blashandle);

    // Free the device memory
    hipFree(dPCsrColPtr);
    hipFree(dPCsrRowPtr);
    hipFree(dPCsrValPtr);
    hipFree(dRCsrColPtr);
    hipFree(dRCsrRowPtr);
    hipFree(dRCsrValPtr);
    hipFree(d_arg_epsilon);
    hipFree(dX);
    hipFree(dY);
    hipFree(dStaticY);
    hipFree(dZ);
    hipFree(dRw);
    hipFree(dRMv);
    hipFree(dRstaticMx);
    hipFree(dBuffer);
    hipFree(dBufferR);
    free(epsilon);
    
}

}