#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

/*
First we need to check that the device can perform upstream and downstream copies
simultaneously.
*/

/*
#######################################################################
#                              CUDA                                   #
#######################################################################
*/

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("cuSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUBLAS(func)                                                     \
{                                                                              \
    hipblasStatus_t status = (func);                                            \
    if (status != HIPBLAS_STATUS_SUCCESS) {                                     \
        printf("CUBLAS API failed at line %d with error: %d\n",                \
               __LINE__, status);                                              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

extern "C" {

// Do C interface stuff
int check_device_properties(void) {
    hipDeviceProp_t GPUProp;

    CHECK_CUDA(hipGetDeviceProperties(&GPUProp, 0));

    printf("Simultaneous transfer = %s\n", GPUProp.deviceOverlap ? "YES" : "NO");
    return 0;
}

// Test a function  which starts a stream and then turns a the
// three input arrays into a CSR array on the stream
int test_stream_csr(
    int *pi, 
    int *pj, 
    float* px, 
    int* pm, 
    int* pn, 
    int* pnz,
    int fsizem,
    int fsizen,
    int fnz
    ) {
    hipEvent_t     start, stop;
    float           elaspsedTime;
    // Start the timers
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));
    CHECK_CUDA(hipEventRecord(start, 0));

    // initialise the stream
    hipStream_t stream;
    CHECK_CUDA(hipStreamCreate(&stream));

    // Allocate the memory to the GPU for the steam size
    int *dPCsrRowPtr, *dPCsrColPtr;
    float *dPCsrValPtr;

    // allocate device memory to store the sparse CSR 
    CHECK_CUDA(hipMalloc((void **)&dPCsrValPtr, sizeof(float) * pnz[0]));
    CHECK_CUDA(hipMalloc((void **)&dPCsrRowPtr, sizeof(int) * pm[0]));
    CHECK_CUDA(hipMalloc((void **)&dPCsrColPtr, sizeof(int) * pnz[0]));

    // Allocate the page-locked memory used in the stream
    //CHECK_CUDA(hipHostAlloc((void**)&host_a, fsizem * sizeof(int), hipHostMallocDefault));
    //CHECK_CUDA(hipHostAlloc((void**)&host_))
    //CHECK_CUDA(hipHostAlloc((void**)&host_))

    return 0;
}

}